#include "hip/hip_runtime.h"
#include /* implements */ <cuda/odf_field.h>

#include <chrono>
#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/extrema.h>

#include <cush.h>
#include <vector_ops.h>

namespace pli
{
void create_odfs(
  const uint3&    dimensions       ,
  const unsigned  coefficient_count,
  const float*    coefficients     ,
  const uint2&    tessellations    ,
  const float3&   spacing          ,
  const uint3&    block_size       ,
  const float     scale            ,
        float3*   points           ,
        float4*   colors           ,
        unsigned* indices          )
{
  auto total_start = std::chrono::system_clock::now();
  
  auto dimension_count    = dimensions.z > 1 ? 3 : 2;
  auto tree_max_depth     = log(dimensions.x) / log(2);
  auto tree_voxel_count   = (pow(2, dimension_count * (tree_max_depth + 1.0)) - 1.0) / (pow(2, dimension_count) - 1.0);
  // TODO!

  auto voxel_count        = dimensions.x * dimensions.y * dimensions.z;
  auto tessellation_count = tessellations.x * tessellations.y;
  auto coefficients_size  = voxel_count * coefficient_count ;
  auto point_count        = voxel_count * tessellation_count;

  std::cout << "Allocating and copying spherical harmonics coefficients." << std::endl;
  thrust::device_vector<float> coefficient_vectors(coefficients_size);
  copy_n(coefficients, coefficients_size, coefficient_vectors.begin());
  auto coefficients_ptr = raw_pointer_cast(&coefficient_vectors[0]);

  std::cout << "Sampling sums of spherical harmonics coefficients." << std::endl;
  cush::sample_sums<<<dim3(dimensions), 1>>>(
    dimensions       ,
    coefficient_count,
    tessellations    ,
    coefficients_ptr , 
    points           , 
    indices          );
  hipDeviceSynchronize();

  std::cout << "Converting points to Cartesian coordinates." << std::endl;
  thrust::transform(
    thrust::device,
    points,
    points + point_count,
    points,
    [] COMMON (const float3& point)
    {
      return cush::to_cartesian_coords(point);
    });
  hipDeviceSynchronize();
  
  std::cout << "Normalizing points." << std::endl;
  for (auto i = 0; i < voxel_count; i++)
  {
    float3* max_sample = thrust::max_element(
      thrust::device,
      points +  i      * tessellation_count,
      points + (i + 1) * tessellation_count,
      [] COMMON (const float3& lhs, const float3& rhs)
      {
        return length(lhs) < length(rhs);
      });
  
    thrust::transform(
      thrust::device,
      points +  i      * tessellation_count,
      points + (i + 1) * tessellation_count,
      points +  i      * tessellation_count,
      [max_sample] COMMON(float3 point)
      {
        auto max_sample_length = length(*max_sample);
        point.x /= max_sample_length;
        point.y /= max_sample_length;
        point.z /= max_sample_length;
        return point;
      });
  }
  hipDeviceSynchronize();

  std::cout << "Assigning colors." << std::endl;
  thrust::transform(
    thrust::device,
    points,
    points + point_count,
    colors,
    [] COMMON (const float3& point)
    {
      return make_float4(abs(point.x), abs(point.y), abs(point.z), 1.0);
    });
  hipDeviceSynchronize();

  std::cout << "Translating and scaling points." << std::endl;
  float3 offset       = {
    spacing.x * (block_size.x - 1) * 0.5,
    spacing.y * (block_size.y - 1) * 0.5,
    spacing.z * (block_size.z - 1) * 0.5};
  float3 real_spacing = {
    spacing.x * block_size.x,
    spacing.y * block_size.y,
    spacing.z * block_size.z};
  auto   real_scale   = scale * real_spacing.x * 0.5;
  thrust::transform(
    thrust::device,
    points,
    points + point_count,
    points,
    [=] COMMON (const float3& point)
    {
      auto output = real_scale * point;
      auto index  = int((&point - points) / tessellation_count);
      output.x += offset.x + real_spacing.x * (index / (dimensions.z * dimensions.y));
      output.y += offset.y + real_spacing.y * (index /  dimensions.z % dimensions.y);
      output.z += offset.z + real_spacing.z * (index % dimensions.z);
      return output;
    });
  hipDeviceSynchronize();

  auto total_end = std::chrono::system_clock::now();
  std::chrono::duration<double> total_elapsed_seconds = total_end - total_start;
  std::cout << "Total elapsed time: " << total_elapsed_seconds.count() << "s." << std::endl;
}
}
