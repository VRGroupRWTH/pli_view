#include "hip/hip_runtime.h"
#include /* implements */ <cuda/vector_field.h>

#include <chrono>
#include <iostream>

#include <thrust/device_vector.h>

namespace pli
{
void create_vector_field(
  const uint3&  dimensions  ,
  const float*  directions  ,
  const float*  inclinations,
  const float&  scale       ,
  const float3& spacing     ,
        float3* points      ,
        float4* colors      )
{
  auto total_start = std::chrono::system_clock::now();

  std::cout << "Allocating and copying directions and inclinations." << std::endl;
  auto voxel_count = dimensions.x * dimensions.y * dimensions.z;
  thrust::device_vector<float> directions_vector  (voxel_count);
  thrust::device_vector<float> inclinations_vector(voxel_count);
  copy_n(directions  , voxel_count, directions_vector  .begin());
  copy_n(inclinations, voxel_count, inclinations_vector.begin());
  auto directions_ptr   = raw_pointer_cast(&directions_vector  [0]);
  auto inclinations_ptr = raw_pointer_cast(&inclinations_vector[0]);
  
  std::cout << "Creating vectors." << std::endl;
  create_vectors<<<dim3(dimensions), 1>>>(
    dimensions      , 
    directions_ptr  , 
    inclinations_ptr,
    scale           , 
    spacing         , 
    points          ,
    colors          );
  hipDeviceSynchronize();

  auto total_end = std::chrono::system_clock::now();
  std::chrono::duration<double> total_elapsed_seconds = total_end - total_start;
  std::cout << "Total elapsed time: " << total_elapsed_seconds.count() << "s." << std::endl;
}
}
