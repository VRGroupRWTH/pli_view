#include "hip/hip_runtime.h"
#include <pli_vis/cuda/zernike/launch.h>

#include <hipblas.h>
#include <hipsolver.h>

#include <pli_vis/cuda/zernike/disk.h>
#include <pli_vis/cuda/zernike/zernike.h>
#include "pli_vis/cuda/sh/spherical_harmonics.h"

namespace zer
{
thrust::device_vector<float> pseudoinverse(
  const uint2&                  size, 
  thrust::device_vector<float>& data)
{
  hipblasHandle_t     cublas  ;
  hipsolverHandle_t cusolver;
  hipsolverDnCreate(&cusolver);
  hipblasCreate    (&cublas  );

  int buffer_size;
  hipsolverDnSgesvd_bufferSize(cusolver, size.x, size.y, &buffer_size);
  hipDeviceSynchronize      ();
  auto complex_buffer_size = static_cast<float>(buffer_size);
  
  thrust::device_vector<float> buffer (buffer_size, 0.0);
  thrust::device_vector<int>   info   (1);
  thrust::device_vector<float> u      (size.x * size.x);
  thrust::device_vector<float> e      (size.y);
  thrust::device_vector<float> vt     (size.y * size.y);
  thrust::device_vector<float> ut     (size.x * size.x);
  thrust::device_vector<float> ei_ut  (size.x * size.y);
  thrust::device_vector<float> v_ei_ut(size.x * size.y);
  auto alpha = 1.0F;
  auto beta  = 0.0F;
  
  hipsolverDnSgesvd(
    cusolver            ,
    'A'                 ,
    'A'                 ,
    size.x              ,
    size.y              ,
    data.data().get()   ,
    size.x              ,
    e.data().get()      ,
    u.data().get()      ,
    size.x              ,
    vt.data().get()     ,
    size.y              ,
    buffer.data().get() ,
    buffer_size         ,
    &complex_buffer_size,
    info.data().get()   );
  hipDeviceSynchronize();
  buffer.clear();

  hipblasSgeam(
    cublas         ,
    HIPBLAS_OP_T    ,
    HIPBLAS_OP_N    ,
    size.x         ,
    size.x         ,
    &alpha         ,
    u.data().get() ,
    size.x         ,
    &beta          ,
    nullptr        ,
    size.x         ,
    ut.data().get(),
    size.x         );
  hipDeviceSynchronize();
  u.clear();
  
  thrust::transform(
    e.begin(),
    e.end  (),
    e.begin(),
    [] __host__ __device__(float& entry) -> float
    {
      if (int(entry) == 0)
        return 0;
      return entry = 1.0F / entry;
    });
  hipDeviceSynchronize();

  hipblasSdgmm(
    cublas            ,
    HIPBLAS_SIDE_LEFT  ,
    size.y            ,
    size.x            ,
    ut.data().get()   ,
    size.x            ,
    e.data().get()    ,
    1                 ,
    ei_ut.data().get(),
    size.y            );
  hipDeviceSynchronize();
  ut.clear();
  e .clear();

  hipblasSgemm(
    cublas              ,
    HIPBLAS_OP_T         ,
    HIPBLAS_OP_N         ,
    size.y              ,
    size.x              ,            
    size.y              ,
    &alpha              ,
    vt.data().get()     ,
    size.y              ,
    ei_ut.data().get()  ,
    size.y              ,
    &beta               ,
    v_ei_ut.data().get(),
    size.y              );
  hipDeviceSynchronize();
  vt   .clear();
  ei_ut.clear();

  hipsolverDnDestroy(cusolver);
  hipblasDestroy    (cublas  );

  return v_ei_ut;
}

thrust::device_vector<float> launch(
  const thrust::device_vector<float3>& vectors        ,
  const uint2&                         superpixel_size,
  const uint2&                         disk_partitions,
  const unsigned                       maximum_degree ,
  const bool                           symmetric      )
{
  const auto superpixel_count  = vectors.size() / (superpixel_size.x * superpixel_size.y);
  const auto sample_count      = disk_partitions.x * disk_partitions.y;
  const auto coefficient_count = expansion_size(maximum_degree);

  // Sample a unit disk.
  thrust::device_vector<float2> disk_samples(sample_count);
  sample_disk<<<grid_size_2d(dim3(superpixel_size.x, superpixel_size.y)), block_size_2d()>>>(
    disk_partitions           , 
    disk_samples.data().get());

  // Compute Zernike basis for the unit disk.
  thrust::device_vector<float> basis_matrix(sample_count * coefficient_count);
  compute_basis<<<grid_size_2d(dim3(sample_count, coefficient_count)), block_size_2d()>>>(
    sample_count              , 
    disk_samples.data().get() , 
    coefficient_count         ,
    basis_matrix.data().get());

  // Compute the inverse of the basis matrix.
  auto inverse = pseudoinverse({sample_count, coefficient_count}, basis_matrix);

  // Project the vectors within each superpixel to the unit disk (interpret as a e.g. hexagon), then
  // multiply the resulting vector with the inverse of the basis matrix to obtain the coefficients.
  thrust::device_vector<float> coefficients(superpixel_count * coefficient_count);
  // TODO.
  return coefficients;
}
}
