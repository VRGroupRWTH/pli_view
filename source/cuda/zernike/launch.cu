#include "hip/hip_runtime.h"
#include <pli_vis/cuda/zernike/launch.h>

#include <hipblas.h>
#include <hipsolver.h>

#include <pli_vis/cuda/utility/vector_ops.h>
#include <pli_vis/cuda/zernike/disk.h>
#include <pli_vis/cuda/zernike/zernike.h>

namespace zer
{
__host__ thrust::device_vector<float> pseudoinverse(
  const uint2&                  size, 
  thrust::device_vector<float>& data)
{
  hipblasHandle_t     cublas  ;
  hipsolverHandle_t cusolver;
  hipsolverDnCreate(&cusolver);
  hipblasCreate    (&cublas  );

  int buffer_size;
  hipsolverDnSgesvd_bufferSize(cusolver, size.x, size.y, &buffer_size);
  hipDeviceSynchronize      ();
  auto complex_buffer_size = static_cast<float>(buffer_size);
  
  thrust::device_vector<float> buffer (buffer_size, 0.0);
  thrust::device_vector<int>   info   (1);
  thrust::device_vector<float> u      (size.x * size.x);
  thrust::device_vector<float> e      (size.y);
  thrust::device_vector<float> vt     (size.y * size.y);
  thrust::device_vector<float> ut     (size.x * size.x);
  thrust::device_vector<float> ei_ut  (size.x * size.y);
  thrust::device_vector<float> v_ei_ut(size.x * size.y);
  auto alpha = 1.0F;
  auto beta  = 0.0F;
  
  hipsolverDnSgesvd(
    cusolver            ,
    'A'                 ,
    'A'                 ,
    size.x              ,
    size.y              ,
    data.data().get()   ,
    size.x              ,
    e.data().get()      ,
    u.data().get()      ,
    size.x              ,
    vt.data().get()     ,
    size.y              ,
    buffer.data().get() ,
    buffer_size         ,
    &complex_buffer_size,
    info.data().get()   );
  hipDeviceSynchronize();
  buffer.clear();

  hipblasSgeam(
    cublas         ,
    HIPBLAS_OP_T    ,
    HIPBLAS_OP_N    ,
    size.x         ,
    size.x         ,
    &alpha         ,
    u.data().get() ,
    size.x         ,
    &beta          ,
    nullptr        ,
    size.x         ,
    ut.data().get(),
    size.x         );
  hipDeviceSynchronize();
  u.clear();
  
  thrust::transform(
    e.begin(),
    e.end  (),
    e.begin(),
    [] __host__ __device__(float& entry) -> float
    {
      if (int(entry) == 0)
        return 0;
      return entry = 1.0F / entry;
    });
  hipDeviceSynchronize();

  hipblasSdgmm(
    cublas            ,
    HIPBLAS_SIDE_LEFT  ,
    size.y            ,
    size.x            ,
    ut.data().get()   ,
    size.x            ,
    e.data().get()    ,
    1                 ,
    ei_ut.data().get(),
    size.y            );
  hipDeviceSynchronize();
  ut.clear();
  e .clear();

  hipblasSgemm(
    cublas              ,
    HIPBLAS_OP_T         ,
    HIPBLAS_OP_N         ,
    size.y              ,
    size.x              ,            
    size.y              ,
    &alpha              ,
    vt.data().get()     ,
    size.y              ,
    ei_ut.data().get()  ,
    size.y              ,
    &beta               ,
    v_ei_ut.data().get(),
    size.y              );
  hipDeviceSynchronize();
  vt   .clear();
  ei_ut.clear();

  hipsolverDnDestroy(cusolver);
  hipblasDestroy    (cublas  );

  return v_ei_ut;
}

__global__ void accumulate(
  const uint2   vectors_size   ,
  const float3* vectors        ,
  const uint2   disk_partitions,
  const float2* disk_samples   ,
  const uint2   superpixel_size,
        float*  intermediates  )
{
  const auto x = blockIdx.x * blockDim.x + threadIdx.x;
  const auto y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= vectors_size.x || y >= vectors_size.y)
    return;

  // Place the vector in the center of a unit disk, project it, and scale by superpixel radius.
  auto       vector       = vectors[y + vectors_size.y * x];
  vector.x = cos(vector.z) * (max(superpixel_size.x, superpixel_size.y) / 2.0F);
  vector.z = M_PI / 2;
  
  // Find the closest sample to the projected endpoint of the vector and accumulate it.
  const auto superpixel_x        = x / superpixel_size.x;
  const auto superpixel_y        = y / superpixel_size.y;
  const auto superpixel_index    = superpixel_y + superpixel_size.y * superpixel_x;
  const auto intermediate_offset = disk_partitions.x * disk_partitions.y * superpixel_index;
  atomicAdd(&intermediates[intermediate_offset + 42], 1);
}

thrust::device_vector<float> launch(
  const thrust::device_vector<float3>& vectors        ,
  const uint2&                         vectors_size   ,
  const uint2&                         superpixel_size,
  const uint2&                         disk_partitions,
  const unsigned                       maximum_degree )
{
  const auto superpixel_count  = vectors.size() / (superpixel_size.x * superpixel_size.y);
  const auto sample_count      = disk_partitions.x * disk_partitions.y;
  const auto coefficient_count = expansion_size(maximum_degree);

  // Sample a unit disk.
  thrust::device_vector<float2> disk_samples(sample_count);
  sample_disk<<<grid_size_2d(dim3(disk_partitions.x, disk_partitions.y)), block_size_2d()>>>(
    disk_partitions           , 
    disk_samples.data().get());

  // Compute Zernike basis for the samples.
  thrust::device_vector<float> basis_matrix(sample_count * coefficient_count);
  compute_basis<<<grid_size_2d(dim3(sample_count, coefficient_count)), block_size_2d()>>>(
    sample_count              , 
    disk_samples.data().get() , 
    coefficient_count         ,
    basis_matrix.data().get());

  // Compute the inverse of the basis matrix.
  auto inverse_basis_matrix = pseudoinverse({sample_count, coefficient_count}, basis_matrix);
  
  // First pass: Accumulate.
  thrust::device_vector<float> intermediates(superpixel_count * sample_count);
  accumulate<<<grid_size_2d(dim3(vectors_size.x, vectors_size.y)), block_size_2d()>>> (
    vectors_size              ,
    vectors      .data().get(),
    disk_partitions           ,
    disk_samples .data().get(),
    superpixel_size           ,
    intermediates.data().get());

  // Second pass: Project.
  thrust::device_vector<float> coefficients(superpixel_count * coefficient_count);
  // TODO!

  return coefficients;
}
}
