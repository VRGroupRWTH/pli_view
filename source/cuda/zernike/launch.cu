#include "hip/hip_runtime.h"
#include <pli_vis/cuda/zernike/launch.h>

#include <hipblas.h>
#include <hipsolver.h>

#include <pli_vis/cuda/zernike/disk.h>
#include <pli_vis/cuda/zernike/zernike.h>
#include "pli_vis/cuda/sh/spherical_harmonics.h"

namespace zer
{
// Matrix pseudoinverse through SVD. Generalize and move to a more accessible place.
__host__ thrust::device_vector<float> pseudoinverse(
  const uint2&                  size, 
  thrust::device_vector<float>& data)
{
  hipblasHandle_t     cublas  ;
  hipsolverHandle_t cusolver;
  hipsolverDnCreate(&cusolver);
  hipblasCreate    (&cublas  );

  int buffer_size;
  hipsolverDnSgesvd_bufferSize(cusolver, size.x, size.y, &buffer_size);
  hipDeviceSynchronize      ();
  auto complex_buffer_size = static_cast<float>(buffer_size);
  
  thrust::device_vector<float> buffer (buffer_size, 0.0);
  thrust::device_vector<int>   info   (1);
  thrust::device_vector<float> u      (size.x * size.x);
  thrust::device_vector<float> e      (size.y);
  thrust::device_vector<float> vt     (size.y * size.y);
  thrust::device_vector<float> ut     (size.x * size.x);
  thrust::device_vector<float> ei_ut  (size.x * size.y);
  thrust::device_vector<float> v_ei_ut(size.x * size.y);
  auto alpha = 1.0F;
  auto beta  = 0.0F;
  
  hipsolverDnSgesvd(
    cusolver            ,
    'A'                 ,
    'A'                 ,
    size.x              ,
    size.y              ,
    data.data().get()   ,
    size.x              ,
    e.data().get()      ,
    u.data().get()      ,
    size.x              ,
    vt.data().get()     ,
    size.y              ,
    buffer.data().get() ,
    buffer_size         ,
    &complex_buffer_size,
    info.data().get()   );
  hipDeviceSynchronize();
  buffer.clear();

  hipblasSgeam(
    cublas         ,
    HIPBLAS_OP_T    ,
    HIPBLAS_OP_N    ,
    size.x         ,
    size.x         ,
    &alpha         ,
    u.data().get() ,
    size.x         ,
    &beta          ,
    nullptr        ,
    size.x         ,
    ut.data().get(),
    size.x         );
  hipDeviceSynchronize();
  u.clear();
  
  thrust::transform(
    e.begin(),
    e.end  (),
    e.begin(),
    [] __host__ __device__(float& entry) -> float
    {
      if (int(entry) == 0)
        return 0;
      return entry = 1.0F / entry;
    });
  hipDeviceSynchronize();

  hipblasSdgmm(
    cublas            ,
    HIPBLAS_SIDE_LEFT  ,
    size.y            ,
    size.x            ,
    ut.data().get()   ,
    size.x            ,
    e.data().get()    ,
    1                 ,
    ei_ut.data().get(),
    size.y            );
  hipDeviceSynchronize();
  ut.clear();
  e .clear();

  hipblasSgemm(
    cublas              ,
    HIPBLAS_OP_T         ,
    HIPBLAS_OP_N         ,
    size.y              ,
    size.x              ,            
    size.y              ,
    &alpha              ,
    vt.data().get()     ,
    size.y              ,
    ei_ut.data().get()  ,
    size.y              ,
    &beta               ,
    v_ei_ut.data().get(),
    size.y              );
  hipDeviceSynchronize();
  vt   .clear();
  ei_ut.clear();

  hipsolverDnDestroy(cusolver);
  hipblasDestroy    (cublas  );

  return v_ei_ut;
}

// Retrieve which superpixel hexagon a pixel is in.
// Based on Troyseph's answer at https://stackoverflow.com/questions/7705228/hexagonal-grids-how-do-you-find-which-hexagon-a-point-is-in
__device__ __host__ uint2 hexagon_id(const uint2& coordinates, const uint2& superpixel_size)
{
  const uint2 grid_size   = {superpixel_size.x, unsigned(0.75 * superpixel_size.y)};
  const auto  half_width  = grid_size.x / 2;
  const auto  y_intercept = unsigned(0.25 * superpixel_size.y);
  const auto  slope       = y_intercept / half_width;

  const auto    row_is_odd = coordinates.y / grid_size.y % 2 == 1;
  uint2         id          {coordinates.y / grid_size.y, (row_is_odd ? coordinates.x - half_width : coordinates.x) / grid_size.x};
  const double2 relative    {coordinates.x - id.y * grid_size.x - (row_is_odd ? half_width : 0), coordinates.y - id.x * grid_size.y};
 
  if      (relative.y < -slope * relative.x + y_intercept)
  {
    id.x--;
    if (!row_is_odd) id.y--;
  }
  else if (relative.y <  slope * relative.x - y_intercept)
  {
    id.x--;
    if (row_is_odd) id.y++;
  }
  return id;
}

// First pass to accumulate vectors into the samples.
__global__ void accumulate(
  const uint2&  vectors_size   ,
  const float3* vectors        ,
  const uint2&  disk_partitions,
  const 
  )
{
  // Retrieve the closest sample within the superpixel hexagon a pixel is in.

}

thrust::device_vector<float> launch(
  const thrust::device_vector<float3>& vectors        ,
  const uint2&                         vectors_size   ,
  const uint2&                         superpixel_size,
  const uint2&                         disk_partitions,
  const unsigned                       maximum_degree )
{
  const auto superpixel_count  = vectors.size() / (superpixel_size.x * superpixel_size.y);
  const auto sample_count      = disk_partitions.x * disk_partitions.y;
  const auto coefficient_count = expansion_size(maximum_degree);

  // Sample a unit disk.
  thrust::device_vector<float2> disk_samples(sample_count);
  sample_disk<<<grid_size_2d(dim3(disk_partitions.x, disk_partitions.y)), block_size_2d()>>>(
    disk_partitions           , 
    disk_samples.data().get());

  // Compute Zernike basis for the samples.
  thrust::device_vector<float> basis_matrix(sample_count * coefficient_count);
  compute_basis<<<grid_size_2d(dim3(sample_count, coefficient_count)), block_size_2d()>>>(
    sample_count              , 
    disk_samples.data().get() , 
    coefficient_count         ,
    basis_matrix.data().get());

  // Compute the inverse of the basis matrix.
  auto inverse_basis_matrix = pseudoinverse({sample_count, coefficient_count}, basis_matrix);
  
  // First pass.
  // For each vector:
  // - Find the superpixel it lies in.
  // - Place the vector in the center of a unit disk and project it to the plane of the disk.
  // - Scale the sampled point's distance to center by the supervoxel (hexagon) size.
  // - Find the closest sample to the projected endpoint of the vector and accumulate it.
  thrust::device_vector<float> intermediates(superpixel_count * sample_count);
  accumulate<<<grid_size_2d(dim3(vectors_size.x, vectors_size.y)), block_size_2d()>>> (
    vectors_size              ,
    vectors      .data().get(),
    disk_partitions           ,
    disk_samples .data().get(),
    superpixel_count          ,
    intermediates.data().get());

  // Second pass.
  // For each hexagon:
  // - Multiply the samples histogram with the inverse of the basis matrix, leading to the coefficients.
  thrust::device_vector<float> coefficients(superpixel_count * coefficient_count);

  return coefficients;
}
}
