#include "hip/hip_runtime.h"
#include <pli_vis/cuda/zernike/launch.h>

#include <hipblas.h>
#include <hipsolver.h>

#include <pli_vis/cuda/utility/vector_ops.h>
#include <pli_vis/cuda/zernike/disk.h>
#include <pli_vis/cuda/zernike/zernike.h>

namespace zer
{
__host__ thrust::device_vector<float> pseudoinverse(const uint2& size, thrust::device_vector<float>& data)
{
  hipblasHandle_t     cublas  ;
  hipsolverHandle_t cusolver;
  hipsolverDnCreate(&cusolver);
  hipblasCreate    (&cublas  );

  int buffer_size;
  hipsolverDnSgesvd_bufferSize(cusolver, size.x, size.y, &buffer_size);
  hipDeviceSynchronize      ();
  auto complex_buffer_size = static_cast<float>(buffer_size);
  
  thrust::device_vector<float> buffer (buffer_size, 0.0);
  thrust::device_vector<int>   info   (1);
  thrust::device_vector<float> u      (size.x * size.x);
  thrust::device_vector<float> e      (size.y);
  thrust::device_vector<float> vt     (size.y * size.y);
  thrust::device_vector<float> ut     (size.x * size.x);
  thrust::device_vector<float> ei_ut  (size.x * size.y);
  thrust::device_vector<float> v_ei_ut(size.x * size.y);
  auto alpha = 1.0F;
  auto beta  = 0.0F;
  
  hipsolverDnSgesvd(
    cusolver            ,
    'A'                 ,
    'A'                 ,
    size.x              ,
    size.y              ,
    data.data().get()   ,
    size.x              ,
    e.data().get()      ,
    u.data().get()      ,
    size.x              ,
    vt.data().get()     ,
    size.y              ,
    buffer.data().get() ,
    buffer_size         ,
    &complex_buffer_size,
    info.data().get()   );
  hipDeviceSynchronize();
  buffer.clear();

  hipblasSgeam(
    cublas         ,
    HIPBLAS_OP_T    ,
    HIPBLAS_OP_N    ,
    size.x         ,
    size.x         ,
    &alpha         ,
    u.data().get() ,
    size.x         ,
    &beta          ,
    nullptr        ,
    size.x         ,
    ut.data().get(),
    size.x         );
  hipDeviceSynchronize();
  u.clear();
  
  thrust::transform(
    e.begin(),
    e.end  (),
    e.begin(),
    [] __host__ __device__(float& entry) -> float
    {
      if (int(entry) == 0)
        return 0;
      return entry = 1.0F / entry;
    });
  hipDeviceSynchronize();

  hipblasSdgmm(
    cublas            ,
    HIPBLAS_SIDE_LEFT  ,
    size.y            ,
    size.x            ,
    ut.data().get()   ,
    size.x            ,
    e.data().get()    ,
    1                 ,
    ei_ut.data().get(),
    size.y            );
  hipDeviceSynchronize();
  ut.clear();
  e .clear();

  hipblasSgemm(
    cublas              ,
    HIPBLAS_OP_T         ,
    HIPBLAS_OP_N         ,
    size.y              ,
    size.x              ,            
    size.y              ,
    &alpha              ,
    vt.data().get()     ,
    size.y              ,
    ei_ut.data().get()  ,
    size.y              ,
    &beta               ,
    v_ei_ut.data().get(),
    size.y              );
  hipDeviceSynchronize();
  vt   .clear();
  ei_ut.clear();

  hipsolverDnDestroy(cusolver);
  hipblasDestroy    (cublas  );

  return v_ei_ut;
}

__global__ void accumulate(
  const uint2    vectors_size         ,
  const float3*  vectors              ,
  const uint2    disk_partitions      ,
  const float2*  disk_samples         ,
  const uint2    superpixel_size      ,
  const uint2    superpixel_dimensions,
        float*   intermediates        )
{
  const auto x = blockIdx.x * blockDim.x + threadIdx.x;
  const auto y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= vectors_size.x || y >= vectors_size.y)
    return;

  const auto& vector = vectors[y + vectors_size.y * x];

  const auto superpixel_x        = x / superpixel_size.x;
  const auto superpixel_y        = y / superpixel_size.y;
  const auto superpixel_index    = superpixel_y + superpixel_dimensions.y * superpixel_x;
  const auto sample_count        = disk_partitions.x * disk_partitions.y;
  const auto intermediate_offset = sample_count * superpixel_index;

  auto min_distance = 2.0F;
  auto sample_index = 0;
  for(auto i = 0; i < sample_count; i++)
  {
    const auto temp_distance = sqrt(pow(cos(vector.z), 2) + pow(disk_samples[i].x, 2) - 2.0F * cos(vector.z) * disk_samples[i].x * cos(vector.y - disk_samples[i].y));
    if (temp_distance < min_distance)
    {
      min_distance = temp_distance;
      sample_index = i;
    }
  }

  atomicAdd(&intermediates[intermediate_offset + sample_index], 1.0F);
}

__global__ void project(
  const uint2    dimensions           ,
  const float*   basis_matrix         ,
  const unsigned sample_count         ,
  const float*   intermediates        ,
  const unsigned coefficient_count    ,
        float*   coefficients         )
{
  const auto x = blockIdx.x * blockDim.x + threadIdx.x;
  const auto y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= dimensions.x || y >= dimensions.y)
    return;

  const auto linear_index        = y + dimensions.y * x;
  const auto intermediate_offset = sample_count      * linear_index;
  const auto coefficient_offset  = coefficient_count * linear_index;

  hipblasHandle_t cublas;
  hipblasCreate(&cublas);
  auto alpha = 1.0F;
  auto beta  = 0.0F;
  hipblasSgemv(
    cublas                             ,
    HIPBLAS_OP_N                        ,
    coefficient_count                  ,
    sample_count                       ,
    &alpha                             ,
    basis_matrix                       ,
    coefficient_count                  ,
    &intermediates[intermediate_offset],
    1                                  ,
    &beta                              ,
    &coefficients [coefficient_offset ],
    1                                  );
  hipblasDestroy(cublas);
}

thrust::device_vector<float> launch(
  const thrust::device_vector<float3>& vectors        ,
  const uint2&                         vectors_size   ,
  const uint2&                         superpixel_size,
  const uint2&                         disk_partitions,
  const unsigned                       maximum_degree )
{
  const auto superpixel_dimensions = uint2{vectors_size.x / superpixel_size.x, vectors_size.y / superpixel_size.y};
  const auto superpixel_count      = superpixel_dimensions.x * superpixel_dimensions.y;
  const auto sample_count          = disk_partitions.x * disk_partitions.y;
  const auto coefficient_count     = expansion_size(maximum_degree);

  // Sample a unit disk.
  thrust::device_vector<float2> disk_samples(sample_count);
  sample_disk<<<grid_size_2d(dim3(disk_partitions.x, disk_partitions.y)), block_size_2d()>>>(
    disk_partitions           , 
    disk_samples.data().get());

  // Compute Zernike basis for the samples.
  thrust::device_vector<float> basis_matrix(sample_count * coefficient_count);
  compute_basis<<<grid_size_2d(dim3(sample_count, coefficient_count)), block_size_2d()>>>(
    sample_count              , 
    disk_samples.data().get() , 
    coefficient_count         ,
    basis_matrix.data().get());

  // Compute the inverse of the basis matrix.
  auto inverse_basis_matrix = pseudoinverse({sample_count, coefficient_count}, basis_matrix);
  
  // Accumulate vectors into superpixels.
  thrust::device_vector<float> intermediates(superpixel_count * sample_count);
  accumulate<<<grid_size_2d(dim3(vectors_size.x, vectors_size.y)), block_size_2d()>>> (
    vectors_size              ,
    vectors      .data().get(),
    disk_partitions           ,
    disk_samples .data().get(),
    superpixel_size           ,
    superpixel_dimensions     ,
    intermediates.data().get());

  // Project superpixels to the Zernike basis.
  thrust::device_vector<float> coefficients(superpixel_count * coefficient_count);
  project<<<grid_size_2d(dim3(superpixel_dimensions.x, superpixel_dimensions.y)), block_size_2d()>>> (
    superpixel_dimensions            ,
    inverse_basis_matrix.data().get(),
    sample_count                     ,
    intermediates.data().get()       ,
    coefficient_count                ,
    coefficients.data().get())       ;

  return coefficients;
}
}
